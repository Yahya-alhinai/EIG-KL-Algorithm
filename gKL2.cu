#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <cmath>
#include <sstream>
#include <limits>
#include <chrono>
#include <sys/stat.h>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <thread>
#include <iomanip>

namespace cg = cooperative_groups;

// ---------------------------------------------------------------------
// 1. GLOBALS & CONSTANTS
// ---------------------------------------------------------------------

bool EIG_init = false;
float globalMin = std::numeric_limits<float>::max();
const float HOST_EPSILON = 1e-6f;
const int MAX_POWER_ITERATIONS = 1000;
const float POWER_CONVERGENCE = 1e-6f;

// Device-side constants
__constant__ int d_terminateLimit;
__constant__ float d_epsilon;

// ---------------------------------------------------------------------
// 2. STRUCTURE DEFINITIONS
// ---------------------------------------------------------------------

struct alignas(16) sparseMatrix {
    unsigned int nodeNum;
    
    // Original members remain the same
    std::vector<std::vector<int>> Nodes;
    std::vector<std::vector<float>> Weights;
    std::vector<int> split[2];
    std::vector<int> remain[2];
    alignas(16) std::vector<int> adjacencyOffsets;
    alignas(16) std::vector<int> adjacencyIndices;
    alignas(16) std::vector<float> adjacencyWeights;
    
    // Device pointers for original data
    int* d_adjacencyOffsets = nullptr;
    int* d_adjacencyIndices = nullptr;
    float* d_adjacencyWeights = nullptr;
    
    // New members for sparse Laplacian
    int* d_laplacianOffsets = nullptr;
    int* d_laplacianIndices = nullptr;
    float* d_laplacianValues = nullptr;
    
    // EIG vectors
    float* d_eigenvector = nullptr;
    float* d_tempVector = nullptr;
};

// Sparse matrix-vector multiplication kernel
__global__ void sparseMVKernel(
    const int* __restrict__ rowOffsets,
    const int* __restrict__ colIndices,
    const float* __restrict__ values,
    const float* __restrict__ x,
    float* __restrict__ y,
    int numRows,
    float shift
) {
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < numRows) {
        const int start = rowOffsets[row];
        const int end = rowOffsets[row + 1];
        float sum = 0.0f;
        
        // Compute dot product for this row
        for (int i = start; i < end; i++) {
            sum += values[i] * x[colIndices[i]];
        }
        
        // Apply shift-and-invert transformation
        y[row] = x[row] - sum / shift;
    }
}


// ---------------------------------------------------------------------
// 3. ERROR CHECKING MACRO
// ---------------------------------------------------------------------

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(err); \
    } \
}

// ---------------------------------------------------------------------
// 4. EIG CUDA KERNELS
// ---------------------------------------------------------------------

// Kernel to construct Laplacian matrix
__global__ void constructLaplacianKernel(
    const int* __restrict__ adjacencyOffsets,
    const int* __restrict__ adjacencyIndices,
    const float* __restrict__ adjacencyWeights,
    float* __restrict__ laplacianMatrix,
    int nodeNum
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < nodeNum) {
        float diagSum = 0.0f;
        int start = adjacencyOffsets[row];
        int end = adjacencyOffsets[row + 1];
        
        // Initialize row to zero
        for (int col = 0; col < nodeNum; col++) {
            laplacianMatrix[row * nodeNum + col] = 0.0f;
        }
        
        // Fill off-diagonal elements and compute diagonal sum
        for (int e = start; e < end; e++) {
            int col = adjacencyIndices[e];
            float weight = -2.0f * adjacencyWeights[e] / 
                          static_cast<float>(end - start); // Normalize by degree
            laplacianMatrix[row * nodeNum + col] = weight;
            diagSum -= weight;
        }
        
        // Set diagonal element
        laplacianMatrix[row * nodeNum + row] = diagSum;
    }
}

// Compute norm kernel
__global__ void computeNormKernel(
    const float* __restrict__ vector,
    float* __restrict__ norm,
    int nodeNum
) {
    __shared__ float s_sum[256];
    const int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    
    // Initialize shared memory
    s_sum[tid] = 0.0f;
    
    // Compute partial sums
    while (idx < nodeNum) {
        s_sum[tid] += vector[idx] * vector[idx];
        idx += gridDim.x * blockDim.x;
    }
    __syncthreads();
    
    // Reduction in shared memory
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_sum[tid] += s_sum[tid + stride];
        }
        __syncthreads();
    }
    
    // Write result
    if (tid == 0) {
        atomicAdd(norm, s_sum[0]);
    }
}

// Modified normalize vector kernel
__global__ void normalizeVectorKernel(
    float* __restrict__ vector,
    const float* __restrict__ norm,
    int nodeNum
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float sqrtNorm = sqrtf(*norm);
    
    if (idx < nodeNum && sqrtNorm > 0.0f) {
        vector[idx] /= sqrtNorm;
    }
}

// Modified power iteration kernel
__global__ void powerIterationKernel(
    const float* __restrict__ laplacianMatrix,
    const float* __restrict__ vector,
    float* __restrict__ result,
    float shift,
    int nodeNum
) {
    __shared__ float s_vector[256];
    const int tid = threadIdx.x;
    const int row = blockIdx.x * blockDim.x + tid;
    
    float sum = 0.0f;
    
    // Process row in tiles
    for (int tile = 0; tile < (nodeNum + blockDim.x - 1) / blockDim.x; ++tile) {
        // Load tile of vector into shared memory
        const int tileOffset = tile * blockDim.x;
        if (tileOffset + tid < nodeNum) {
            s_vector[tid] = vector[tileOffset + tid];
        }
        __syncthreads();
        
        // Compute partial sum for this tile
        if (row < nodeNum) {
            const int tileSize = min(blockDim.x, nodeNum - tileOffset);
            for (int j = 0; j < tileSize; ++j) {
                sum += laplacianMatrix[row * nodeNum + (tileOffset + j)] * s_vector[j];
            }
        }
        __syncthreads();
    }
    
    // Write result with shift-and-invert transformation
    if (row < nodeNum) {
        result[row] = vector[row] - sum / shift;
    }
}

void writeEIGResults(const std::string& baseName,
                    const std::vector<float>& eigenvalues,
                    const std::vector<float>& eigenvector,
                    float medianValue,
                    int nodeNum) {
    std::string outfile = "pre_saved_EIG/" + baseName + "_out.txt";
    std::ofstream fout(outfile);
    if (!fout.is_open()) {
        std::cerr << "Error opening output file: " << outfile << std::endl;
        return;
    }
    
    // Write first eigenvalue
    fout << std::setprecision(12) << eigenvalues[0] << std::endl;
    
    // Write median value
    fout << std::setprecision(12) << medianValue << std::endl;
    
    // Write node assignments
    for (int i = 0; i < nodeNum; i++) {
        fout << i << "\t" << (medianValue > eigenvector[i] ? 1 : 0) << "\t"
             << std::setprecision(12) << eigenvector[i] << std::endl;
    }
    
    fout.close();
    std::cout << "EIG results written to: " << outfile << std::endl;
}

void computeEigenpartition(sparseMatrix& spMat) {
    std::cout << "\nComputing EIG partitioning...\n";
    const int nodeNum = spMat.nodeNum;
    
    // First, construct sparse Laplacian matrix in CPU
    std::vector<int> laplacianOffsets(nodeNum + 1, 0);
    std::vector<int> laplacianIndices;
    std::vector<float> laplacianValues;
    
    // Count non-zeros per row first
    #pragma omp parallel for
    for (int i = 0; i < nodeNum; i++) {
        laplacianOffsets[i + 1] = spMat.Nodes[i].size() + 1; // +1 for diagonal
    }
    
    // Compute offsets
    for (int i = 0; i < nodeNum; i++) {
        laplacianOffsets[i + 1] += laplacianOffsets[i];
    }
    
    // Allocate arrays for indices and values
    size_t totalNnz = laplacianOffsets[nodeNum];
    laplacianIndices.resize(totalNnz);
    laplacianValues.resize(totalNnz);
    
    // Fill the sparse Laplacian matrix
    #pragma omp parallel for
    for (int i = 0; i < nodeNum; i++) {
        int offset = laplacianOffsets[i];
        float diagSum = 0.0f;
        
        // Off-diagonal elements
        for (size_t j = 0; j < spMat.Nodes[i].size(); j++) {
            int col = spMat.Nodes[i][j];
            float weight = -2.0f * spMat.Weights[i][j] / 
                          static_cast<float>(spMat.Nodes[i].size());
            
            laplacianIndices[offset] = col;
            laplacianValues[offset] = weight;
            diagSum -= weight;
            offset++;
        }
        
        // Diagonal element
        laplacianIndices[offset] = i;
        laplacianValues[offset] = diagSum;
    }
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc(&spMat.d_laplacianOffsets, (nodeNum + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc(&spMat.d_laplacianIndices, totalNnz * sizeof(int)));
    CHECK_CUDA(hipMalloc(&spMat.d_laplacianValues, totalNnz * sizeof(float)));
    CHECK_CUDA(hipMalloc(&spMat.d_eigenvector, nodeNum * sizeof(float)));
    CHECK_CUDA(hipMalloc(&spMat.d_tempVector, nodeNum * sizeof(float)));
    
    // Copy Laplacian to device
    CHECK_CUDA(hipMemcpy(spMat.d_laplacianOffsets, laplacianOffsets.data(),
                         (nodeNum + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(spMat.d_laplacianIndices, laplacianIndices.data(),
                         totalNnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(spMat.d_laplacianValues, laplacianValues.data(),
                         totalNnz * sizeof(float), hipMemcpyHostToDevice));
    
    // Initialize random eigenvector
    std::vector<float> initialVector(nodeNum);
    srand(42);
    for (int i = 0; i < nodeNum; i++) {
        initialVector[i] = static_cast<float>(rand()) / RAND_MAX - 0.5f;
    }
    CHECK_CUDA(hipMemcpy(spMat.d_eigenvector, initialVector.data(),
                         nodeNum * sizeof(float), hipMemcpyHostToDevice));
    
    float* d_norm;
    CHECK_CUDA(hipMalloc(&d_norm, sizeof(float)));
    
    dim3 blockDim(256);
    dim3 gridDim(min((nodeNum + blockDim.x - 1) / blockDim.x, 65535));
    
    float shift = 2.0f;
    float prevNorm = 0.0f;
    
    // Power iteration
    std::cout << "Starting power iteration...\n";
    for (int iter = 0; iter < MAX_POWER_ITERATIONS; iter++) {
        // Reset norm
        float zero = 0.0f;
        CHECK_CUDA(hipMemcpy(d_norm, &zero, sizeof(float), hipMemcpyHostToDevice));
        
        sparseMVKernel<<<gridDim, blockDim>>>(
            spMat.d_laplacianOffsets,
            spMat.d_laplacianIndices,
            spMat.d_laplacianValues,
            spMat.d_eigenvector,
            spMat.d_tempVector,
            nodeNum,
            shift
        );
        
        // Compute norm
        computeNormKernel<<<gridDim, blockDim>>>(
            spMat.d_tempVector,
            d_norm,
            nodeNum
        );
        
        // Normalize
        normalizeVectorKernel<<<gridDim, blockDim>>>(
            spMat.d_tempVector,
            d_norm,
            nodeNum
        );
        
        // Check convergence
        float norm;
        CHECK_CUDA(hipMemcpy(&norm, d_norm, sizeof(float), hipMemcpyDeviceToHost));
        
        if (std::abs(norm - prevNorm) < POWER_CONVERGENCE && iter > 100) {
            std::cout << "Converged after " << iter << " iterations\n";
            break;
        }
        prevNorm = norm;
        
        // Swap pointers
        float* tempPtr = spMat.d_eigenvector;
        spMat.d_eigenvector = spMat.d_tempVector;
        spMat.d_tempVector = tempPtr;
        
        if (iter % 100 == 0) {
            std::cout << "Iteration " << iter << ", norm diff: " 
                      << std::abs(norm - prevNorm) << std::endl;
        }
    }
    
    // Get final eigenvector
    std::vector<float> eigenvector(nodeNum);
    CHECK_CUDA(hipMemcpy(eigenvector.data(), spMat.d_eigenvector,
                         nodeNum * sizeof(float), hipMemcpyDeviceToHost));
    
    // Compute median and partition
    std::vector<float> sortedEig = eigenvector;
    std::sort(sortedEig.begin(), sortedEig.end());
    float median = sortedEig[nodeNum / 2];
    
    // Create partitions
    spMat.split[0].clear();
    spMat.split[1].clear();
    spMat.remain[0].clear();
    spMat.remain[1].clear();
    
    for (int i = 0; i < nodeNum; i++) {
        if (median > eigenvector[i]) {
            spMat.split[0].push_back(i);
            spMat.remain[0].push_back(i);
        } else {
            spMat.split[1].push_back(i);
            spMat.remain[1].push_back(i);
        }
    }
    
    std::cout << "Initial partition sizes: " << spMat.split[0].size() 
              << " and " << spMat.split[1].size() << "\n";
    
    // Cleanup
    CHECK_CUDA(hipFree(spMat.d_laplacianOffsets));
    CHECK_CUDA(hipFree(spMat.d_laplacianIndices));
    CHECK_CUDA(hipFree(spMat.d_laplacianValues));
    CHECK_CUDA(hipFree(spMat.d_eigenvector));
    CHECK_CUDA(hipFree(spMat.d_tempVector));
    CHECK_CUDA(hipFree(d_norm));
    
    spMat.d_laplacianOffsets = nullptr;
    spMat.d_laplacianIndices = nullptr;
    spMat.d_laplacianValues = nullptr;
    spMat.d_eigenvector = nullptr;
    spMat.d_tempVector = nullptr;
}

void shuffleSparceMatrix(sparseMatrix& spMat) {
    spMat.split[0].clear();
    spMat.split[1].clear();
    spMat.remain[0].clear();
    spMat.remain[1].clear();
    
    if (EIG_init) {
        computeEigenpartition(spMat);
        return;
    }
    
    // Random partitioning if not using EIG
    std::vector<int> all;
    all.reserve(spMat.nodeNum);
    for (unsigned int i = 0; i < spMat.nodeNum; i++) {
        all.push_back(i);
    }
    std::random_shuffle(all.begin(), all.end());
    
    unsigned int half = spMat.nodeNum / 2;
    for (unsigned int i = 0; i < half; i++) {
        spMat.split[0].push_back(all[i]);
        spMat.remain[0].push_back(all[i]);
    }
    for (unsigned int i = half; i < spMat.nodeNum; i++) {
        spMat.split[1].push_back(all[i]);
        spMat.remain[1].push_back(all[i]);
    }
}

// ---------------------------------------------------------------------
// 5. GPU KERNEL: connectionsKernel
//    out[i] = E - I for remain[i],
//    E = external (sum of edges to other partition)
//    I = internal (sum of edges to own partition)
// ---------------------------------------------------------------------

// Define the kernels for the KL Algorithm on GPU
template<int BLOCK_SIZE>
__global__ void connectionsKernel(
    const int* __restrict__ adjacencyOffsets,
    const int* __restrict__ adjacencyIndices,
    const float* __restrict__ adjacencyWeights,
    const int* __restrict__ membership,
    const int* __restrict__ d_remain,
    float* __restrict__ d_out,
    int remainSize
) {
    // Only keeping the memory we actually use
    __shared__ int s_membership[BLOCK_SIZE];  
    
    auto block = cg::this_thread_block();
    const int tid = block.thread_rank();
    const int idx = blockIdx.x * blockDim.x + tid;
    
    if (idx < remainSize) {
        const int node = d_remain[idx];
        const int start = adjacencyOffsets[node];
        const int end = adjacencyOffsets[node + 1];
        
        s_membership[tid] = membership[node];
        block.sync();
        
        float E = 0.f, I = 0.f;
        const int mySide = s_membership[tid];
        
        #pragma unroll 4
        for (int e = start; e < end; e++) {
            const int neigh = adjacencyIndices[e];
            const float w = adjacencyWeights[e];
            if (membership[neigh] == mySide) {
                I += w;
            } else {
                E += w;
            }
        }
        
        d_out[idx] = (E - I);
    }
}

// Copy the adjacency matrix to the device
void copyAdjacencyToDevice(sparseMatrix& spMat, hipStream_t stream) {
    const size_t offsetSize = (spMat.nodeNum + 1) * sizeof(int);
    const size_t totalEdges = spMat.adjacencyOffsets[spMat.nodeNum];
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&spMat.d_adjacencyOffsets, offsetSize));
    CHECK_CUDA(hipMalloc((void**)&spMat.d_adjacencyIndices, totalEdges * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&spMat.d_adjacencyWeights, totalEdges * sizeof(float)));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(spMat.d_adjacencyOffsets, spMat.adjacencyOffsets.data(),
                         offsetSize, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(spMat.d_adjacencyIndices, spMat.adjacencyIndices.data(),
                         totalEdges * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(spMat.d_adjacencyWeights, spMat.adjacencyWeights.data(),
                         totalEdges * sizeof(float), hipMemcpyHostToDevice));
}

void freeDeviceAdjacency(sparseMatrix& spMat, hipStream_t stream) {
    if (spMat.d_adjacencyOffsets) {
        CHECK_CUDA(hipFree(spMat.d_adjacencyOffsets));
        spMat.d_adjacencyOffsets = nullptr;
    }
    if (spMat.d_adjacencyIndices) {
        CHECK_CUDA(hipFree(spMat.d_adjacencyIndices));
        spMat.d_adjacencyIndices = nullptr;
    }
    if (spMat.d_adjacencyWeights) {
        CHECK_CUDA(hipFree(spMat.d_adjacencyWeights));
        spMat.d_adjacencyWeights = nullptr;
    }
}


// ---------------------------------------------------------------------
// 6. GPU CONNECTIONS: single call for remain[] nodes
//    This version reuses allocated buffers d_remain, d_out, d_membership
//    so we only do hipMemcpy each iteration, not hipMalloc/hipFree.
// ---------------------------------------------------------------------

void gpuConnections(
    const sparseMatrix& spMat,
    int* d_remain,
    int* d_membership,
    float* d_out,
    const std::vector<int>& remain,
    const std::vector<int>& membershipHost,
    std::vector<float>& out,
    hipStream_t stream
) {
    const int remainSize = static_cast<int>(remain.size());
    if (remainSize == 0) return;
    
    constexpr int BLOCK_SIZE = 256;
    const int gridSize = (remainSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Copy data (remain, membership) to device
    CHECK_CUDA(hipMemcpyAsync(d_remain, remain.data(),
                              remainSize * sizeof(int),
                              hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_membership, membershipHost.data(),
                              spMat.nodeNum * sizeof(int),
                              hipMemcpyHostToDevice, stream));
    
    // Launch the kernel
    connectionsKernel<BLOCK_SIZE><<<gridSize, BLOCK_SIZE, 0, stream>>>(
        spMat.d_adjacencyOffsets,
        spMat.d_adjacencyIndices,
        spMat.d_adjacencyWeights,
        d_membership,
        d_remain,
        d_out,
        remainSize
    );
    
    // Copy the results back to host
    CHECK_CUDA(hipMemcpyAsync(out.data(), d_out,
                              remainSize * sizeof(float),
                              hipMemcpyDeviceToHost, stream));
}

// ---------------------------------------------------------------------
// 7. HELPER FUNCTIONS
// ---------------------------------------------------------------------

// Helper function to create a directory
void createDir(const std::string& dirName) {
    struct stat info;
    if (stat(dirName.c_str(), &info) != 0) {
        mkdir(dirName.c_str(), 0755);
    }
}

std::string getBaseName(const std::string& path) {
    std::filesystem::path fp(path);
    return fp.filename().string();
}


// Helper function to build the flattened adjacency list
void buildFlattenedAdjacency(sparseMatrix& spMat) {
    spMat.adjacencyOffsets.resize(spMat.nodeNum + 1, 0);
    
    #pragma omp parallel for
    for (unsigned int i = 0; i < spMat.nodeNum; i++) {
        spMat.adjacencyOffsets[i + 1] = spMat.adjacencyOffsets[i] + spMat.Nodes[i].size();
    }
    
    size_t totalEdges = spMat.adjacencyOffsets[spMat.nodeNum];
    spMat.adjacencyIndices.resize(totalEdges);
    spMat.adjacencyWeights.resize(totalEdges);
    
    #pragma omp parallel for
    for (unsigned int i = 0; i < spMat.nodeNum; i++) {
        int start = spMat.adjacencyOffsets[i];
        for (unsigned int j = 0; j < spMat.Nodes[i].size(); j++) {
            spMat.adjacencyIndices[start + j] = spMat.Nodes[i][j];
            spMat.adjacencyWeights[start + j] = spMat.Weights[i][j];
        }
    }
}

float computeCutSize(const sparseMatrix& spMat, const std::vector<int>& membership) {
    float E = 0.f;
    #pragma omp parallel for reduction(+:E)
    for (auto node0 : spMat.split[0]) {
        const auto& nbrs = spMat.Nodes[node0];
        const auto& wts = spMat.Weights[node0];
        for (size_t j = 0; j < nbrs.size(); j++) {
            if (membership[nbrs[j]] == 1) {
                E += wts[j];
            }
        }
    }
    return E;
}

float nodeConnection(const sparseMatrix& spMat, int a, int b) {
    const auto& nbrs = spMat.Nodes[a];
    const auto& wts = spMat.Weights[a];
    for (size_t i = 0; i < nbrs.size(); i++) {
        if (nbrs[i] == b) return wts[i];
    }
    return 0.f;
}

void swip(sparseMatrix& spMat, std::vector<int>& membership, int num1, int num2) {
    membership[num1] = 1;
    membership[num2] = 0;
    
    auto it1 = std::find(spMat.remain[0].begin(), spMat.remain[0].end(), num1);
    if (it1 != spMat.remain[0].end()) spMat.remain[0].erase(it1);
    
    auto it2 = std::find(spMat.remain[1].begin(), spMat.remain[1].end(), num2);
    if (it2 != spMat.remain[1].end()) spMat.remain[1].erase(it2);
    
    auto s1 = std::find(spMat.split[0].begin(), spMat.split[0].end(), num1);
    if (s1 != spMat.split[0].end()) *s1 = num2;
    
    auto s2 = std::find(spMat.split[1].begin(), spMat.split[1].end(), num2);
    if (s2 != spMat.split[1].end()) *s2 = num1;
}

// ---------------------------------------------------------------------
// 8. MAIN KL IMPLEMENTATION
// ---------------------------------------------------------------------

// Add this helper function for periodic verification
float verifyAndCorrectCutSize(const sparseMatrix& spMat, 
                            const std::vector<int>& membership, 
                            float& currentCutSize,
                            int iteration) {
    // Verify every N iterations (e.g., every 10 iterations)
    const int VERIFY_INTERVAL = 10;
    if (iteration % VERIFY_INTERVAL == 0) {
        float verifiedCutSize = computeCutSize(spMat, membership);
        if (std::abs(verifiedCutSize - currentCutSize) > HOST_EPSILON) {
            // If difference detected, correct it
            currentCutSize = verifiedCutSize;
        }
    }
    return currentCutSize;
}

float calculateGain(const sparseMatrix& spMat,
                   const std::vector<int>& membership,
                   int node1, int node2,
                   float maxGain1, float maxGain2) {
    // Calculate edge weight between nodes
    float edgeWeight = nodeConnection(spMat, node1, node2);
    
    // Use Kahan summation for better numerical stability
    float sum = 0.0f;
    float c = 0.0f;  // compensation
    
    // Add maxGain1
    float y = maxGain1 - c;
    float t = sum + y;
    c = (t - sum) - y;
    sum = t;
    
    // Add maxGain2
    y = maxGain2 - c;
    t = sum + y;
    c = (t - sum) - y;
    sum = t;
    
    // Subtract edge weight contribution
    y = -2.0f * edgeWeight - c;
    t = sum + y;
    c = (t - sum) - y;
    sum = t;
    
    return sum;
}


void KL(sparseMatrix& spMat) {
    std::cout << "\n========== Starting Optimized KL Algorithm ==========\n";
    
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    
    shuffleSparceMatrix(spMat);
    std::vector<int> membership(spMat.nodeNum, -1);
    for (auto n : spMat.split[0]) membership[n] = 0;
    for (auto n : spMat.split[1]) membership[n] = 1;
    
    float cutSize = computeCutSize(spMat, membership);
    const float initialCutSize = cutSize;
    float bestCut = cutSize;
    
    const int maxRem = std::max(spMat.remain[0].size(), spMat.remain[1].size());
    int* d_remain;
    float* d_out;
    int* d_membership;
    
    CHECK_CUDA(hipMallocManaged(&d_remain, maxRem * sizeof(int)));
    CHECK_CUDA(hipMallocManaged(&d_membership, spMat.nodeNum * sizeof(int)));
    CHECK_CUDA(hipMallocManaged(&d_out, maxRem * sizeof(float)));
    
    int iteration = 0;
    int terminate = 0;
    const int terminateLimit = static_cast<int>(std::log2(spMat.nodeNum)) + 5;
    auto startTime = std::chrono::high_resolution_clock::now();
    
    std::cout << "\nIteration Progress:\n";
    std::cout << std::setw(10) << "Iteration" 
              << std::setw(15) << "Cut Size" 
              << std::setw(18) << "Gain"
              << std::setw(18) << "Time (ms)"
              << std::setw(16) << "Improvement\n";
    
    while (!spMat.remain[0].empty() && !spMat.remain[1].empty()) {
        auto iterStart = std::chrono::high_resolution_clock::now();
        
        std::vector<float> con_1(spMat.remain[0].size());
        std::vector<float> con_2(spMat.remain[1].size());
        
        gpuConnections(spMat, d_remain, d_membership, d_out,
                    spMat.remain[0], membership, con_1, stream);
        gpuConnections(spMat, d_remain, d_membership, d_out,
                    spMat.remain[1], membership, con_2, stream);
        
        float maxGain1 = -std::numeric_limits<float>::infinity();
        float maxGain2 = -std::numeric_limits<float>::infinity();
        int node1 = -1, node2 = -1;
        
        // Find optimal nodes with stable summation
        for (size_t i = 0; i < con_1.size(); i++) {
            if (con_1[i] > maxGain1) {
                maxGain1 = con_1[i];
                node1 = spMat.remain[0][i];
            }
        }
        
        for (size_t i = 0; i < con_2.size(); i++) {
            if (con_2[i] > maxGain2) {
                maxGain2 = con_2[i];
                node2 = spMat.remain[1][i];
            }
        }
        
        // Calculate gain with improved numerical stability
        float gain = calculateGain(spMat, membership, node1, node2, maxGain1, maxGain2);
        
        // Update cut size with periodic verification
        cutSize -= gain;
        // cutSize = verifyAndCorrectCutSize(spMat, membership, cutSize, iteration);
        bestCut = std::min(bestCut, cutSize);
        
        // Perform swap
        swip(spMat, membership, node1, node2);
        
        // Update termination condition
        if (gain <= HOST_EPSILON) {  // Using host-side epsilon instead of device constant
            if (++terminate > terminateLimit) break;
        } else {
            terminate = 0;
        }
        
        // Update global minimum
        globalMin = std::min(globalMin, cutSize);
        
        if (iteration % 100 == 0) {
            // Print iteration progress every 100 iterations
            auto iterEnd = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(iterEnd - iterStart);
            float improvement = 100.0f * (1.0f - cutSize / initialCutSize);
            
            std::cout << std::setw(8) << iteration 
                  << std::setw(17) << std::fixed << std::setprecision(2) << cutSize 
                  << std::setw(18) << std::fixed << std::setprecision(2) << gain 
                  << std::setw(15) << duration.count()
                  << std::setw(15) << std::fixed << std::setprecision(2) << improvement << "%\n";
        }
        
        iteration++;

    }
    
    // Cleanup and synchronize
    CHECK_CUDA(hipStreamSynchronize(stream));
    
    // Final verification
    float finalCheck = computeCutSize(spMat, membership);
    if (std::abs(finalCheck - cutSize) > HOST_EPSILON) {  // Using host-side epsilon
        std::cout << "\nWarning: Cut size verification difference detected.\n"
                  << "Incremental: " << cutSize << ", From-scratch: " << finalCheck << std::endl;
        cutSize = finalCheck;
    }
    
    // Print final results
    auto endTime = std::chrono::high_resolution_clock::now();
    auto totalDuration = std::chrono::duration_cast<std::chrono::seconds>(endTime - startTime);
    
    std::cout << "\n=============== Final Results =================\n";
    std::cout << "Total iterations: " << iteration << "\n";
    std::cout << "Initial cut size: " << std::fixed << std::setprecision(2) << initialCutSize << "\n";
    std::cout << "Best cut size   : " << bestCut << "\n";
    std::cout << "Improvement     : " << std::fixed << std::setprecision(2) 
              << 100.0f * (1.0f - bestCut/initialCutSize) << "%\n";
    std::cout << "Total runtime   : " << totalDuration.count() << " seconds\n";
    
    // Ensure all operations are complete then free memory
    CHECK_CUDA(hipStreamSynchronize(stream));
    CHECK_CUDA(hipFree(d_remain));
    CHECK_CUDA(hipFree(d_membership));
    CHECK_CUDA(hipFree(d_out));
}

// ---------------------------------------------------------------------
// 9. GPU INFO AND INITIALIZATION
// ---------------------------------------------------------------------

void printGPUInfo() {
    int deviceCount = 0;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));
    
    std::cout << "\n================= GPU Info ===================\n";
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        CHECK_CUDA(hipGetDeviceProperties(&prop, i));
        std::cout << "Device " << i << ": " << prop.name << "\n";
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << "\n";
        std::cout << "  Memory: " << prop.totalGlobalMem / (1024*1024*1024.0) << " GB\n";
        std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << "\n";
        std::cout << "  Max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << "\n";
        std::cout << "  Number of multiprocessors: " << prop.multiProcessorCount << "\n";
        
    }
}

void InitializeSparsMatrix(const std::string& filename, sparseMatrix& spMat) {
    std::cout << "\n============= Reading Input File ==============\n";
    std::ifstream fin(filename);
    if (!fin.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    
    std::string line;
    std::getline(fin, line);
    int netsNum = 0, nodesNum = 0;
    {
        std::stringstream ss(line);
        ss >> netsNum >> nodesNum;
    }
    
    std::cout << "Circuit Statistics\n";
    std::cout << "  - Total Nets : " << netsNum << "\n";
    std::cout << "  - Total Nodes: " << nodesNum << "\n";
    
    spMat.nodeNum = nodesNum;
    spMat.Nodes.resize(nodesNum);
    spMat.Weights.resize(nodesNum);
    
    for (auto& node : spMat.Nodes) {
        node.reserve(32);  // Typical average degree
    }
    for (auto& weight : spMat.Weights) {
        weight.reserve(32);
    }
    
    long int nonZeroElements = nodesNum;
    long int numEdges = 0;
    
    #pragma omp parallel for reduction(+:nonZeroElements,numEdges)
    for (int i = 0; i < netsNum; i++) {
        std::string localLine;
        #pragma omp critical
        {
            std::getline(fin, localLine);
        }
        
        std::stringstream ss(localLine);
        std::vector<int> nodes;
        int nd;
        while (ss >> nd) {
            nodes.push_back(nd);
        }
        
        if (nodes.size() < 2) continue;
        float weight = 1.f / float(nodes.size() - 1);
        
        for (size_t j = 0; j < nodes.size(); j++) {
            for (size_t k = j + 1; k < nodes.size(); k++) {
                numEdges++;
                int a = nodes[j] - 1, b = nodes[k] - 1;
                
                #pragma omp critical
                {
                    auto it = std::find(spMat.Nodes[a].begin(), spMat.Nodes[a].end(), b);
                    if (it == spMat.Nodes[a].end()) {
                        spMat.Nodes[a].push_back(b);
                        spMat.Weights[a].push_back(weight);
                        spMat.Nodes[b].push_back(a);
                        spMat.Weights[b].push_back(weight);
                        nonZeroElements += 2;
                    } else {
                        int idxA = it - spMat.Nodes[a].begin();
                        spMat.Weights[a][idxA] += weight;
                        auto it2 = std::find(spMat.Nodes[b].begin(), spMat.Nodes[b].end(), a);
                        int idxB = it2 - spMat.Nodes[b].begin();
                        spMat.Weights[b][idxB] += weight;
                    }
                }
            }
        }
    }
    fin.close();
    
    buildFlattenedAdjacency(spMat);
    
    float fullMatrixSize = static_cast<float>(nodesNum * nodesNum * sizeof(float)) / (1024.0f * 1024.0f);
    float sparseMatrixSize = static_cast<float>(nonZeroElements * (sizeof(float) + 2 * sizeof(int))) / (1024.0f * 1024.0f);
    
    std::cout << "\n============= Matrix Statistics ===============\n";
    std::cout << "Matrix Dimensions\n";
    std::cout << "  - Full matrix: " << nodesNum << " x " << nodesNum << "\n";
    std::cout << "  - Non-zero   : " << nonZeroElements << "\n";
    std::cout << "  - Density    : " << std::fixed << std::setprecision(3)
              << (100.0f * nonZeroElements / (static_cast<uint64_t>(nodesNum) * nodesNum)) << "%\n";
    std::cout << "\nMemory Usage\n";
    std::cout << "  - Full matrix  : " << std::fixed << std::setprecision(3) << fullMatrixSize << " MB\n";
    std::cout << "  - Sparse matrix: " << std::fixed << std::setprecision(3) << sparseMatrixSize << " MB\n";
}


// ---------------------------------------------------------------------
// 7. MAIN FUNCTION
// ---------------------------------------------------------------------

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <inputFile> [-EIG]\n";
        return 1;
    }
    
    createDir("results");
    createDir("pre_saved_EIG");
    
    std::string inputFile = argv[1];
    std::string baseName = getBaseName(inputFile);
    
    if (argc == 3 && std::string(argv[2]) == "-EIG") {
        EIG_init = true;
    }
    
    std::string foutName = "results/" + baseName;
    foutName += EIG_init ? "_KL_CutSize_EIG_output.txt" : "_KL_CutSize_output.txt";
    
    printGPUInfo();
    
    sparseMatrix spMat;
    InitializeSparsMatrix(inputFile, spMat);
    
    hipStream_t mainStream;
    CHECK_CUDA(hipStreamCreate(&mainStream));
    
    copyAdjacencyToDevice(spMat, mainStream);
    
    if (EIG_init) {
        computeEigenpartition(spMat);
    }
    
    std::this_thread::sleep_for(std::chrono::seconds(3));
    
    KL(spMat);
    
    freeDeviceAdjacency(spMat, mainStream);
    CHECK_CUDA(hipStreamDestroy(mainStream));
    
    std::cout << "\nGlobal Minimum Cut across all runs: " << std::fixed 
              << std::setprecision(2) << globalMin << std::endl;
    
    return 0;
}